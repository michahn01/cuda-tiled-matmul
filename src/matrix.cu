#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <cstring>
#include <cassert>

#include <vector>
#include <stdexcept>

#include <hip/hip_runtime.h>

#include "matrix.h"

#define TILE_WIDTH 16
#define EPSILON 0.00001

#define CUDA_ASSERT(res) ( check_cuda_error((res), __FILE__, __LINE__) )
inline void check_cuda_error(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA check failed in file %s line %d: %s", file, line, hipGetErrorString(code));
        exit(code);
    }
}


/* -------------------------------------------------------------------- */
/* -------------------- MATMUL: CPU IMPLEMENTATION -------------------- */
/* -------------------------------------------------------------------- */

/*
Use the CPU to perform matrix multiplication on pointers to 1D row-major
matrix representations of floats.

Assumes dimensions match and `output` points to well-defined memory.
*/
void cpu_matmul(
    float* A, float* B, float* output,
    uint32_t A_rows, uint32_t A_cols,
    uint32_t B_rows, uint32_t B_cols
) {
    for (int i = 0; i < A_rows; ++i) {
        for (int j = 0; j < B_cols; ++j) {
            float entry = 0;
            for (int k = 0; k < A_cols; ++k) {
                entry += A[i * A_cols + k] * B[k * B_cols + j];
            }
            output[i * B_cols + j] = entry;
        }
    }
}


/* -------------------------------------------------------------------- */
/* ---------------- MATMUL: UNTILED GPU IMPLEMENTATION ---------------- */
/* -------------------------------------------------------------------- */

/* Note: TILED version of GPU matmul is also implemented further down this file. */

__global__
void untiled_gpu_matmul_kernel(
    float* A, float* B, float* output,
    uint32_t A_rows, uint32_t A_cols,
    uint32_t B_rows, uint32_t B_cols
) {
    uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols) {
        float entry = 0;
        for (int k = 0; k < A_cols; ++k) {
            entry += A[row * A_cols + k] * B[k * B_cols + col];
        }
        output[row * B_cols + col] = entry;
    }
}

/*
Use the GPU to perform matrix multiplication on pointers to 1D row-major
matrix representations of floats. Does not utilize tiling. 

Assumes dimensions match and `output` points to well-defined memory.
*/
void untiled_gpu_matmul(
    float* A_h, float* B_h, float* output_h,
    uint32_t A_rows, uint32_t A_cols,
    uint32_t B_rows, uint32_t B_cols
) {
    uint32_t A_size = A_rows * A_cols * sizeof(float);
    uint32_t B_size = B_rows * B_cols * sizeof(float);
    uint32_t output_size = A_rows * B_cols * sizeof(float);

    // Allocate memory on device (GPU).
    float* A_d;
    float* B_d;
    float* output_d;
    hipMalloc((void**)&A_d, A_size);
    hipMalloc((void**)&B_d, B_size);
    hipMalloc((void**)&output_d, output_size);

    // Copy matrices from host to device.
    hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

    // Configure dimensions and launch kernel.
    dim3 block_dim(32, 32);
    dim3 grid_dim(ceil(B_cols / 32.0), ceil(A_rows / 32.0));
    untiled_gpu_matmul_kernel<<<grid_dim, block_dim>>>(
        A_d, B_d, output_d,
        A_rows, A_cols, B_rows, B_cols
    );

    // Check that kernel launch was successful.
    GPU_ASSERT(hipGetLastError());

    // Copy answer from device to host.
    GPU_ASSERT(hipMemcpy(output_h, output_d, output_size, hipMemcpyDeviceToHost));

    // Free memory and return.
    hipFree(A_d);
    hipFree(B_d);
    hipFree(output_d);
}


/* -------------------------------------------------------------------- */
/* ----------------- MATMUL: TILED GPU IMPLEMENTATION ----------------- */
/* -------------------------------------------------------------------- */

__global__
void gpu_matmul_kernel(
    float* A, float* B, float* output,
    uint32_t A_rows, uint32_t A_cols,
    uint32_t B_rows, uint32_t B_cols
) {
    extern __shared__ float Ads[];
    float* Bds = Ads + TILE_WIDTH * TILE_WIDTH;

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int tile_idx = ty * TILE_WIDTH + tx;

    uint32_t row = by * blockDim.y + ty;
    uint32_t col = bx * blockDim.x + tx;

    float entry = 0;

    // Repeatedly process tiles across A_cols (equal to B_rows).
    for (int tile = 0; tile < ceil(float(A_cols) / TILE_WIDTH); ++tile) {
        // (PHASE 1) Collaboratively load data into shared memory pool.
        if (row < A_rows && tile * TILE_WIDTH + tx < A_cols) {
            Ads[tile_idx] = A[row * A_cols + tile * TILE_WIDTH + tx];
        }
        else {
            Ads[tile_idx] = 0;
        }
        if (col < B_cols && tile * TILE_WIDTH + ty < B_rows) {
            Bds[tile_idx] = B[col + (tile * TILE_WIDTH + ty) * B_cols];
        }
        else {
            Bds[tile_idx] = 0;
        }

        // Wait for all threads to finish filling up shared memory.
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            // Ads[ty][k] * Bds[k][tx]
            entry += (
                Ads[ty * TILE_WIDTH + k] *
                Bds[k * TILE_WIDTH + tx]
            );
        }

        // Wait for all threads to finish reading this round of shared memory.
        __syncthreads();
    }

    if (row < A_rows && col < B_cols) {
        output[row * B_cols + col] = entry;
    }
}

/*
Use the GPU to perform matrix multiplication on pointers to 1D row-major
matrix representations of floats. Utilizes tiling.

Assumes dimensions match and `output` points to well-defined memory.
*/
void gpu_matmul(
    float* A_h, float* B_h, float* output_h,
    uint32_t A_rows, uint32_t A_cols,
    uint32_t B_rows, uint32_t B_cols
) {

    uint32_t A_size = A_rows * A_cols * sizeof(float);
    uint32_t B_size = B_rows * B_cols * sizeof(float);
    uint32_t output_size = A_rows * B_cols * sizeof(float);

    // Allocate memory on device (GPU).
    float* A_d;
    float* B_d;
    float* output_d;
    hipMalloc((void**)&A_d, A_size);
    hipMalloc((void**)&B_d, B_size);
    hipMalloc((void**)&output_d, output_size);

    // Copy matrices from host to device.
    hipMemcpy(A_d, A_h, A_size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, B_size, hipMemcpyHostToDevice);

    // Configure dimensions and launch kernel.
    dim3 block_dim(TILE_WIDTH, TILE_WIDTH);
    dim3 grid_dim(
        ceil(double(B_cols) / TILE_WIDTH),
        ceil(double(A_rows) / TILE_WIDTH)
    );

    int shared_mem_bytes = 2 * TILE_WIDTH * TILE_WIDTH * sizeof(float);
    gpu_matmul_kernel<<<grid_dim, block_dim, shared_mem_bytes>>>(
        A_d, B_d, output_d,
        A_rows, A_cols, B_rows, B_cols
    );

    // Check that kernel launch was successful.
    GPU_ASSERT(hipGetLastError());

    // Copy answer from device to host.
    GPU_ASSERT(hipMemcpy(output_h, output_d, output_size, hipMemcpyDeviceToHost));

    // Free memory and return.
    hipFree(A_d);
    hipFree(B_d);
    hipFree(output_d);
}
